#include "hip/hip_runtime.h"
#include "preprocess.h"

__global__
void cuda_resize(const uchar4* d_rawImg,
                 int rows,int cols,
                 uchar4* d_resizeImg)  //1024*1280 --> 512*640
{
    int2 idx_2d=make_int2((blockIdx.x*blockDim.x)+threadIdx.x,(blockIdx.y*blockDim.y)+threadIdx.y);
    int idx_1d=cols*idx_2d.y+idx_2d.x;
    if(idx_2d.x>=cols||idx_2d.y>=rows)
        return;
    if(idx_2d.x%2==0&&idx_2d.y%2==0)
    {
        int2 res_idx_2d=make_int2(idx_2d.x/2,idx_2d.y/2);
        int res_idx_1d=res_idx_2d.y*cols/2+res_idx_2d.x;

        d_resizeImg[res_idx_1d]=d_rawImg[idx_1d];
    }

}

__global__
void channelComp(const uchar4* input_rgba,
                      int rows,int cols, unsigned char threthold, bool is_tgt_red,
                      unsigned char* redChannel,
                      unsigned char* greenChannel,
                      unsigned char* blueChannel,
                      unsigned char* d_monoImg, // compare result
                      unsigned char* d_compImg //bianry after threth
                      )
{
      int2 idx_2d=make_int2((blockIdx.x*blockDim.x)+threadIdx.x,(blockIdx.y*blockDim.y)+threadIdx.y);
      int idx_1d=cols*idx_2d.y+idx_2d.x;
      if(idx_2d.x>=cols||idx_2d.y>=rows)
          return;
      uchar4 rgba_pix=input_rgba[idx_1d];
      redChannel[idx_1d]=rgba_pix.x;
      greenChannel[idx_1d]=rgba_pix.y;
      blueChannel[idx_1d]=rgba_pix.z;

      //compare red and blue channel
      int comp_pix;
      if(is_tgt_red)        //red-blue
      {
          comp_pix=redChannel[idx_1d]-blueChannel[idx_1d];

      }else
      {
          comp_pix=blueChannel[idx_1d]-redChannel[idx_1d];
      }

      d_monoImg[idx_1d]=comp_pix>0?(unsigned char)comp_pix:0;

      //threath mono to binary
       d_compImg[idx_1d]=(d_monoImg[idx_1d]>threthold)?255:0;



}
void preKernelWrapper(uchar4* d_rawImg, uchar4* d_resizeImg,
                      int rows,int cols, unsigned char threthold, bool is_tgt_red,
                      unsigned char* redChannel,
                      unsigned char* greenChannel,
                      unsigned char* blueChannel,
                      unsigned char* d_monoImg, // compare result
                      unsigned char* d_compImg //bianry after threth
                      )
{
    if(rows==1024&&cols==1280)
    {
        const dim3 blockSize(32,16);
        const dim3 gridSize(1+cols/blockSize.x,1+rows/blockSize.y);
        cuda_resize<<<gridSize,blockSize>>>(d_rawImg,rows,cols,d_resizeImg);

        //update new rows and cols
        rows=512;
        cols=640;
        //update ptr
        d_rawImg=d_resizeImg;
    }

    const dim3 blockSize_split(32,16);
    const dim3 gridSize_split(1+cols/blockSize_split.x,1+rows/blockSize_split.y);

    channelComp<<<gridSize_split,blockSize_split>>>(d_rawImg,
                                                    rows,cols,threthold,is_tgt_red,
                                                    redChannel,
                                                    greenChannel,
                                                    blueChannel,
                                                    d_monoImg,
                                                    d_compImg
                                                    );


}
